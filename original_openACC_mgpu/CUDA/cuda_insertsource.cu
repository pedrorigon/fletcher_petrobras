#include "hip/hip_runtime.h"
#include "cuda_defines.h"
#include "cuda_insertsource.h"

__global__ void kernel_InsertSource(const float val, const int iSource,
	                            float * restrict qp, float * restrict qc)
{
  const int ix=blockIdx.x * blockDim.x + threadIdx.x;
  if (ix==0)
  {
    qp[iSource]+=val;
    qc[iSource]+=val;
  }
}


void CUDA_InsertSource(const float val, const int iSource, float *p, float *q)
{

  extern float* dev_pp;
  extern float* dev_pc;
  extern float* dev_qp;
  extern float* dev_qc;

	
  if ((dev_pp) && (dev_qp))
  {
     dim3 threadsPerBlock(BSIZE_X, 1);
     dim3 numBlocks(1,1);
  
     kernel_InsertSource<<<numBlocks, threadsPerBlock>>> (val, iSource, dev_pc, dev_qc);
     CUDA_CALL(hipGetLastError());
     CUDA_CALL(hipDeviceSynchronize());
  }
}
