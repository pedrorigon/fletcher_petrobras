#include "hip/hip_runtime.h"
#include "cuda_defines.h"
#include "cuda_propagate.h"
#include "../derivatives.h"
#include "../map.h"

__global__ void kernel_Propagate(const int sx, const int sy, const int sz, const int bord,
				 const float dx, const float dy, const float dz, const float dt,  
				 const int it, const float * const restrict ch1dxx, 
				 const float * const restrict ch1dyy, float * restrict ch1dzz, 
				 float * restrict ch1dxy, float * restrict ch1dyz, float * restrict ch1dxz, 
				 float * restrict v2px, float * restrict v2pz, float * restrict v2sz, 
				 float * restrict v2pn, float * restrict pp, float * restrict pc, 
				 float * restrict qp, float * restrict qc)
{
  const int ix=blockIdx.x * blockDim.x + threadIdx.x;
  const int iy=blockIdx.y * blockDim.y + threadIdx.y;

#define SAMPLE_PRE_LOOP
#include "../sample.h"
#undef SAMPLE_PRE_LOOP

    // solve both equations in all internal grid points, 
    // including absortion zone
    
    for (int iz=bord+1; iz<sz-bord-1; iz++) {

#define SAMPLE_LOOP
#include "../sample.h"
#undef SAMPLE_LOOP

    }
}


// Propagate: using Fletcher's equations, propagate waves one dt,
//            either forward or backward in time
void CUDA_Propagate(const int sx, const int sy, const int sz, const int bord,
		    const float dx, const float dy, const float dz, const float dt, const int it, 
		    float * restrict pp, float * restrict pc, float * restrict qp, float * restrict qc)
{
  
  extern float* dev_vpz;
  extern float* dev_vsv;
  extern float* dev_epsilon;
  extern float* dev_delta;
  extern float* dev_phi;
  extern float* dev_theta;
  extern float* dev_ch1dxx;
  extern float* dev_ch1dyy;
  extern float* dev_ch1dzz;
  extern float* dev_ch1dxy;
  extern float* dev_ch1dyz;
  extern float* dev_ch1dxz;
  extern float* dev_v2px;
  extern float* dev_v2pz;
  extern float* dev_v2sz;
  extern float* dev_v2pn;
  extern float* dev_pp;
  extern float* dev_pc;
  extern float* dev_qp;
  extern float* dev_qc;
  extern float* dev_pDx;
  extern float* dev_pDy;
  extern float* dev_qDx;
  extern float* dev_qDy;
  
  
  dim3 threadsPerBlock(BSIZE_X, BSIZE_Y);
  dim3 numBlocks(sx/threadsPerBlock.x, sy/threadsPerBlock.y);
  
  kernel_Propagate<<<numBlocks, threadsPerBlock>>> (  sx,   sy,   sz,   bord,
						      dx,   dy,   dz,   dt,   it, 
						      dev_ch1dxx,  dev_ch1dyy,  dev_ch1dzz, 
						      dev_ch1dxy,  dev_ch1dyz,  dev_ch1dxz, 
						      dev_v2px,  dev_v2pz,  dev_v2sz,  dev_v2pn,
						      dev_pp,  dev_pc,  dev_qp,  dev_qc);
  CUDA_CALL(hipGetLastError());
  CUDA_SwapArrays(&dev_pp, &dev_pc, &dev_qp, &dev_qc);
  CUDA_CALL(hipDeviceSynchronize());
}

// swap array pointers on time forward array propagation
void CUDA_SwapArrays(float **pp, float **pc, float **qp, float **qc) {
  float *tmp;
  
  tmp=*pp;
  *pp=*pc;
  *pc=tmp;
  
  tmp=*qp;
  *qp=*qc;
  *qc=tmp;
}
