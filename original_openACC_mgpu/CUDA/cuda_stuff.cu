#include "cuda_defines.h"
#include "cuda_stuff.h"

static size_t sxsy=0;

void CUDA_Initialize(const int sx, const int sy, const int sz, const int bord,
	       float dx, float dy, float dz, float dt,
	       float * restrict ch1dxx, float * restrict ch1dyy, float * restrict ch1dzz, 
	       float * restrict ch1dxy, float * restrict ch1dyz, float * restrict ch1dxz, 
	       float * restrict v2px, float * restrict v2pz, float * restrict v2sz, float * restrict v2pn,
	       float * restrict vpz, float * restrict vsv, float * restrict epsilon, float * restrict delta,
	       float * restrict phi, float * restrict theta, 
	       float * restrict pp, float * restrict pc, float * restrict qp, float * restrict qc)
{

   extern float* dev_pDx;
   extern float* dev_pDy;
   extern float* dev_qDx;
   extern float* dev_qDy;
   extern float* dev_vpz;
   extern float* dev_vsv;
   extern float* dev_epsilon;
   extern float* dev_delta;
   extern float* dev_phi;
   extern float* dev_theta;
   extern float* dev_ch1dxx;
   extern float* dev_ch1dyy;
   extern float* dev_ch1dzz;
   extern float* dev_ch1dxy;
   extern float* dev_ch1dyz;
   extern float* dev_ch1dxz;
   extern float* dev_v2px;
   extern float* dev_v2pz;
   extern float* dev_v2sz;
   extern float* dev_v2pn;
   extern float* dev_pp;
   extern float* dev_pc;
   extern float* dev_qp;
   extern float* dev_qc;

 
  int deviceCount;
  CUDA_CALL(hipGetDeviceCount(&deviceCount));
  const int device=0;
  hipDeviceProp_t deviceProp;
  CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
  printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
  CUDA_CALL(hipSetDevice(device));


  // Check sx,sy values
  if (sx%BSIZE_X != 0)
  {
     printf("sx(%d) must be multiple of BSIZE_X(%d)\n", sx, (int)BSIZE_X);
     exit(1);
  } 
  if (sy%BSIZE_Y != 0)
  {
     printf("sy(%d) must be multiple of BSIZE_Y(%d)\n", sy, (int)BSIZE_Y);
     exit(1);
  } 

   sxsy=sx*sy; // one plan
   const size_t sxsysz=sxsy*sz;
   const size_t msize_vol=sxsysz*sizeof(float);
   const size_t msize_vol_extra=msize_vol+2*sxsy*sizeof(float); // 2 extra plans for wave fields
   CUDA_CALL(hipMalloc(&dev_vpz, msize_vol));
   CUDA_CALL(hipMemcpy(dev_vpz, vpz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_vsv, msize_vol));
   CUDA_CALL(hipMemcpy(dev_vsv, vsv, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_epsilon, msize_vol));
   CUDA_CALL(hipMemcpy(dev_epsilon, epsilon, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_delta, msize_vol));
   CUDA_CALL(hipMemcpy(dev_delta, delta, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_phi, msize_vol));
   CUDA_CALL(hipMemcpy(dev_phi, phi, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_theta, msize_vol));
   CUDA_CALL(hipMemcpy(dev_theta, theta, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_ch1dxx, msize_vol));
   CUDA_CALL(hipMemcpy(dev_ch1dxx, ch1dxx, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_ch1dyy, msize_vol));
   CUDA_CALL(hipMemcpy(dev_ch1dyy, ch1dyy, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_ch1dzz, msize_vol));
   CUDA_CALL(hipMemcpy(dev_ch1dzz, ch1dzz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_ch1dxy, msize_vol));
   CUDA_CALL(hipMemcpy(dev_ch1dxy, ch1dxy, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_ch1dyz, msize_vol));
   CUDA_CALL(hipMemcpy(dev_ch1dyz, ch1dyz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_ch1dxz, msize_vol));
   CUDA_CALL(hipMemcpy(dev_ch1dxz, ch1dxz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_v2px, msize_vol));
   CUDA_CALL(hipMemcpy(dev_v2px, v2px, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_v2pz, msize_vol));
   CUDA_CALL(hipMemcpy(dev_v2pz, v2pz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_v2sz, msize_vol));
   CUDA_CALL(hipMemcpy(dev_v2sz, v2sz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMalloc(&dev_v2pn, msize_vol));
   CUDA_CALL(hipMemcpy(dev_v2pn, v2pn, msize_vol, hipMemcpyHostToDevice));

   // Wave field arrays with an extra plan
   CUDA_CALL(hipMalloc(&dev_pp, msize_vol_extra));
   CUDA_CALL(hipMemset(dev_pp, 0, msize_vol_extra));
   CUDA_CALL(hipMalloc(&dev_pc, msize_vol_extra));
   CUDA_CALL(hipMemset(dev_pc, 0, msize_vol_extra));
   CUDA_CALL(hipMalloc(&dev_qp, msize_vol_extra));
   CUDA_CALL(hipMemset(dev_qp, 0, msize_vol_extra));
   CUDA_CALL(hipMalloc(&dev_qc, msize_vol_extra));
   CUDA_CALL(hipMemset(dev_qc, 0, msize_vol_extra));
   dev_pp+=sxsy;
   dev_pc+=sxsy;
   dev_qp+=sxsy;
   dev_qc+=sxsy;

   CUDA_CALL(hipMalloc(&dev_pDx, msize_vol));
   CUDA_CALL(hipMemset(dev_pDx, 0, msize_vol));
   CUDA_CALL(hipMalloc(&dev_pDy, msize_vol));
   CUDA_CALL(hipMemset(dev_pDy, 0, msize_vol));
   CUDA_CALL(hipMalloc(&dev_qDx, msize_vol));
   CUDA_CALL(hipMemset(dev_qDx, 0, msize_vol));
   CUDA_CALL(hipMalloc(&dev_qDy, msize_vol));
   CUDA_CALL(hipMemset(dev_qDy, 0, msize_vol));



  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
  printf("GPU memory usage = %ld MiB\n", 21*msize_vol/1024/1024);

}


void CUDA_Finalize()
{

   extern float* dev_vpz;
   extern float* dev_vsv;
   extern float* dev_epsilon;
   extern float* dev_delta;
   extern float* dev_phi;
   extern float* dev_theta;
   extern float* dev_ch1dxx;
   extern float* dev_ch1dyy;
   extern float* dev_ch1dzz;
   extern float* dev_ch1dxy;
   extern float* dev_ch1dyz;
   extern float* dev_ch1dxz;
   extern float* dev_v2px;
   extern float* dev_v2pz;
   extern float* dev_v2sz;
   extern float* dev_v2pn;
   extern float* dev_pp;
   extern float* dev_pc;
   extern float* dev_qp;
   extern float* dev_qc;
   extern float* dev_pDx;
   extern float* dev_pDy;
   extern float* dev_qDx;
   extern float* dev_qDy;

   dev_pp-=sxsy;
   dev_pc-=sxsy;
   dev_qp-=sxsy;
   dev_qc-=sxsy;

   CUDA_CALL(hipFree(dev_vpz));
   CUDA_CALL(hipFree(dev_vsv));
   CUDA_CALL(hipFree(dev_epsilon));
   CUDA_CALL(hipFree(dev_delta));
   CUDA_CALL(hipFree(dev_phi));
   CUDA_CALL(hipFree(dev_theta));
   CUDA_CALL(hipFree(dev_ch1dxx));
   CUDA_CALL(hipFree(dev_ch1dyy));
   CUDA_CALL(hipFree(dev_ch1dzz));
   CUDA_CALL(hipFree(dev_ch1dxy));
   CUDA_CALL(hipFree(dev_ch1dyz));
   CUDA_CALL(hipFree(dev_ch1dxz));
   CUDA_CALL(hipFree(dev_v2px));
   CUDA_CALL(hipFree(dev_v2pz));
   CUDA_CALL(hipFree(dev_v2sz));
   CUDA_CALL(hipFree(dev_v2pn));
   CUDA_CALL(hipFree(dev_pp));
   CUDA_CALL(hipFree(dev_pc));
   CUDA_CALL(hipFree(dev_qp));
   CUDA_CALL(hipFree(dev_qc));
   CUDA_CALL(hipFree(dev_pDx));
   CUDA_CALL(hipFree(dev_qDx));
   CUDA_CALL(hipFree(dev_pDy));
   CUDA_CALL(hipFree(dev_qDy));

   printf("CUDA_Finalize: SUCCESS\n");
}



void CUDA_Update_pointers(const int sx, const int sy, const int sz, float *pc)
{
   extern float* dev_pc;
   const size_t sxsysz=((size_t)sx*sy)*sz;
   const size_t msize_vol=sxsysz*sizeof(float);
   if (pc) CUDA_CALL(hipMemcpy(pc, dev_pc, msize_vol, hipMemcpyDeviceToHost));
}
